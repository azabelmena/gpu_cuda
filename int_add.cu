#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

const int SIZE = 1 << 30;

// Add two integers and store it into a third integer.
__global__ void int_add(int *a, int *b, int *c){

    int i = threadIdx.x;

    if(i < SIZE){
        c[0] = a[0]+b[0];
    }

    return;
}

int main(){

    int a = 5;
    int b = 9;
    int c = 0;

    int *d_a = 0;
    int *d_c = 0;
    int *d_b = 0;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice);

    int_add<<<64, 64>>>(d_a,d_b,d_c);

    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d+%d=%d\n",a,b,c);

    return 0;
}
